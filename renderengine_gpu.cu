#include "hip/hip_runtime.h"
//
// Created by Palash on 11-04-2018.
//

#include "renderengine_gpu.h"
#include "cudaHeaders.h"
#include "world_gpu.h"
#include "camera_gpu.h"
#include "ray_gpu.h"

RenderEngine_GPU::RenderEngine_GPU(World *_world, Camera *_camera) : RenderEngine(_world, _camera) {

}

bool RenderEngine_GPU::renderLoop() {

    static int i = 0;
    static int steps = 0;

    hipEvent_t begin, begin_kernel, stop_kernel, stop;
    hipEventCreate(&begin);
    hipEventCreate(&begin_kernel);
    hipEventCreate(&stop_kernel);
    hipEventCreate(&stop);

    //init vars
    unsigned char *bitmap_gpu;
    hipMalloc(reinterpret_cast<void**>(&bitmap_gpu), IMAGE_HEIGHT * IMAGE_WIDTH * 3 * sizeof(unsigned char));
    Camera_GPU cam(camera);
    World_GPU wor(world);

    hipEventRecord(begin);

    //DO copy all variables
    hipMemcpy(bitmap_gpu, camera->getBitmap(), IMAGE_HEIGHT * IMAGE_WIDTH * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 threadsperblock(SAMPLE,SAMPLE,MAX_THREADS_IN_BLOCK/(SAMPLE*SAMPLE));
    dim3 blockspergrid(IMAGE_HEIGHT * COLUMNS_IN_ONCE/threadsperblock.z);

    hipEventRecord(begin_kernel);
    Main_Render_Kernel << < blockspergrid, threadsperblock >> >(i, bitmap_gpu, cam, wor, steps, rand());
    hipEventRecord(stop_kernel);
    gpuErrchk(hipPeekAtLastError());

    //Copy all variables back
    hipMemcpy(camera->getBitmap(), bitmap_gpu, IMAGE_HEIGHT * IMAGE_WIDTH * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop_kernel);
    hipEventSynchronize(stop);

    float kernelTime, totalTime; // Initialize elapsedTime;
    hipEventElapsedTime(&kernelTime, begin_kernel, stop_kernel);
    hipEventElapsedTime(&totalTime, begin, stop);

    printf("Time: %fms, %fms\n", kernelTime, totalTime);

    //Free variables
    hipFree(bitmap_gpu);

    if( (i+=COLUMNS_IN_ONCE) == camera->getWidth())
    {
        i = 0;
        steps++;
        camera->incSteps();
//        std::cout<<"Samples Done: "<<camera->getSteps()*SAMPLE*SAMPLE<<std::endl;
        return false;
    }
    return false;
}

__device__ float3 computeColor(Ray_GPU ray, unsigned int &seed, World_GPU wor) {
    float3 c = AMBIENT_COLOR;

    unsigned char sphere = wor.intersectRay(ray);
    int loop_end = 0;
    for (unsigned char i = 0; i < MAX_LEVEL; i++){
        if(loop_end){
            continue;
        }
        else if(sphere<wor.n) {
            c = c*wor.spheres[sphere].col;
            SPHERE_MATERIAL sp_mat = wor.spheres[sphere].material;
            if(sp_mat == LIGHT){
                //light
                loop_end = 1;
                continue;
            }else if(sp_mat == DIELECTRIC){
                //dielectric
                float eta = wor.spheres[sphere].param;
                float cosTheta = dot(ray.dir, ray.normal);
                bool isInside = cosTheta > 0;
                float nc=1, nnt=isInside?eta/nc:nc/eta;
                float cos2t = 1-nnt*nnt*(1-cosTheta*cosTheta);
                if (cos2t<0){ //TIR
                    ray.dir = normalize(ray.dir - 2 * ray.normal * cosTheta);
                }else{
                    cosTheta = -fabs(cosTheta);
                    float3 refr_dir = normalize(ray.dir * nnt - ray.normal*((isInside?-1:1)*(cosTheta*nnt+sqrt(cos2t))));

                    float a=eta-nc, b=eta+nc, R0=a*a/(b*b), c1 = 1-(isInside?dot(refr_dir, ray.normal):-cosTheta);
                    float Re=R0+(1-R0)*c1*c1*c1*c1*c1,Tr=1-Re,P=.25f+.5f*Re,RP=Re/P,TP=Tr/(1-P);
                    if (Random_GPU(seed) < P) {
                        c = c * RP;
                        ray.dir = normalize(ray.dir - 2 * ray.normal * cosTheta);
                    }
                    else{
                        c = c * TP;
                        ray.dir = refr_dir;
                    }
                }
            }else if(sp_mat == GLOSSY){
                //glossy
                float cosTheta = dot(ray.dir, ray.normal);
                float n = wor.spheres[sphere].param;

                float phi=2*M_PI*Random_GPU(seed), cosAlpha=pow(Random_GPU(seed), 1.f/(n+1)), sineAlpha = sqrt(1-cosAlpha*cosAlpha);
                float rotAngle = 2*(acos(-cosTheta) + acos(cosAlpha) - M_PI/2);

                float3 w = normalize(ray.dir - 2 * ray.normal * cosTheta);
                float3 u = normalize(cross((fabs(w.x)>.1?make_float3(0,1,0):make_float3(1,0,0)),w));
                float3 v = cross(w,u);

                float3 dDirection = u*cos(phi)*sineAlpha + v*sin(phi)*sineAlpha + w*cosAlpha;

                if(dot(dDirection,ray.normal)<0) {
                    float3 k = normalize(cross(w, ray.normal));
                    dDirection = cos(rotAngle) * dDirection + sin(rotAngle) * cross(k, dDirection);
                }
                ray.dir = dDirection;
            }else if(sp_mat == REFLECTIVE && Random_GPU(seed) < wor.spheres[sphere].param){
                float cosTheta = dot(ray.dir, ray.normal);
                ray.dir = normalize(ray.dir - 2 * ray.normal * cosTheta);
            }else {
                //diffuse
                float alpha=2*M_PI* Random_GPU(seed),z= Random_GPU(seed), sineTheta = sqrtf(1-z);
                float3 w = ray.normal;
                float3 u = normalize(cross((fabs(w.x)>.1?make_float3(0,1,0):make_float3(1,0,0)),w));
                float3 v = cross(w,u);
                ray.dir = u*cos(alpha)*sineTheta + v*sin(alpha)*sineTheta + w*sqrt(z);
            }
            sphere = wor.intersectRay(ray);
        }else{
            c = BACKGROUND;
            loop_end = true;
        }
    }
    return c;
}

__global__ void Main_Render_Kernel(int startI, unsigned char *bitmap, Camera_GPU cam, World_GPU wor, unsigned int steps,
                                   unsigned int mrand) { //j->row, i->column
    // <8,8,12>

    unsigned int p = threadIdx.x;
    unsigned int q = threadIdx.y;

    unsigned int j = (blockIdx.x * blockDim.z + threadIdx.z);
    unsigned int i = startI + j/IMAGE_HEIGHT;
    j %= IMAGE_HEIGHT;

    unsigned int seed = 12345678 + p*11234 + q*23145 + i*13456 + j*14567 + steps*5678 + mrand*49574;
    float _i = i + (p + Random_GPU(seed)) / SAMPLE;
    float _j = j + (q + Random_GPU(seed)) / SAMPLE;


    //Initial Ray direction
    float3 dir = make_float3(0,0,0);
    dir += -cam.w * 1.207107f;
    float xw = (1.0f*IMAGE_WIDTH/IMAGE_HEIGHT * (_i - IMAGE_WIDTH / 2.0f + 0.5f) / IMAGE_WIDTH);
    float yw = ((_j - IMAGE_HEIGHT / 2.0f + 0.5f) / IMAGE_HEIGHT);
    dir += cam.u * xw;
    dir += cam.v * yw;
    dir = normalize(dir);

    //Create ray
    Ray_GPU ray(cam.pos, dir);
    float3 c = computeColor(ray, seed, wor);

    c = warpReduceSumTriple(c);
    __shared__ float3 val[MAX_THREADS_IN_BLOCK/(SAMPLE*SAMPLE)];
    val[threadIdx.z] = make_float3(1,0,0);
    __syncthreads();
    if(p==SAMPLE-1 && q==SAMPLE-1)
        val[threadIdx.z] = c;
    __syncthreads();

    if(p==0 && q==0){
        c = c+val[threadIdx.z];
        c = clamp(c/(SAMPLE*SAMPLE), 0, 1);
        int index = (i + j*IMAGE_WIDTH)*3;
        bitmap[index + 0] = (unsigned char) ((bitmap[index + 0] * steps + 256  * c.x) / (steps + 1));
        bitmap[index + 1] = (unsigned char) ((bitmap[index + 1] * steps + 256 * c.y) / (steps + 1));
        bitmap[index + 2] = (unsigned char) ((bitmap[index + 2] * steps + 256  * c.z) / (steps + 1));
    }
}
