#include "hip/hip_runtime.h"
//
// Created by Palash on 11-04-2018.
//

#include <ctime>
#include "renderengine_gpu.h"
#include "cudaHeaders.h"
#include "world_gpu.h"
#include "camera_gpu.h"
#include "ray_gpu.h"
#include "hiprand/hiprand_kernel.h"

#define MAX_COORD 15
#define ALPHA 0.05f

RenderEngine_GPU::RenderEngine_GPU(World *_world, Camera *_camera) : RenderEngine(_world, _camera), wor(_world), cam(_camera) {
    //init vars
    hipMalloc(reinterpret_cast<void**>(&bitmap_gpu), cam.size.y * cam.size.x * 3 * sizeof(unsigned char));
    hipMalloc(reinterpret_cast<void**>(&random_texture_device), cam.size.y * cam.size.x * sizeof(int));
    hipMalloc(reinterpret_cast<void**>(&q_table_device), MAX_COORD * MAX_COORD * MAX_COORD * 8 * sizeof(float));
    random_texture = (int*)malloc(cam.size.y * cam.size.x * sizeof(int));
    q_table = (float*)malloc(MAX_COORD * MAX_COORD * MAX_COORD * 8 * sizeof(float));
    //Init random texture.
    srand(static_cast<unsigned int>(clock()));
    for(int j = 0; j<cam.size.y * cam.size.x; j++){
        random_texture[j] = rand();
    }
    for(int j = 0; j<MAX_COORD * MAX_COORD * MAX_COORD * 8; j++){
        q_table[j] = .1f*rand()/RAND_MAX;
    }

    //DO copy all variables
    hipMemcpy(random_texture_device, random_texture, cam.size.y * cam.size.x * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(q_table_device, q_table, MAX_COORD * MAX_COORD * MAX_COORD * 8 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(bitmap_gpu, camera->getBitmap(), cam.size.y * cam.size.x * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);
}

bool RenderEngine_GPU::renderLoop() {

    static int i = 0;
    static int steps = 0;

    hipEvent_t begin, begin_kernel, stop_kernel, stop;
    hipEventCreate(&begin);
    hipEventCreate(&begin_kernel);
    hipEventCreate(&stop_kernel);
    hipEventCreate(&stop);

    hipEventRecord(begin);

    //Init random texture.
    srand(static_cast<unsigned int>(clock()));
    for(unsigned int j = 0; j<cam.size.y * cam.size.x; j++){
        random_texture[j] = rand();
    }
    hipMemcpy(random_texture_device, random_texture, cam.size.y * cam.size.x * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsperblock(SAMPLE,SAMPLE,MAX_THREADS_IN_BLOCK/(SAMPLE*SAMPLE));
    dim3 blockspergrid(cam.size.y * COLUMNS_IN_ONCE/threadsperblock.z);

    hipEventRecord(begin_kernel);
    Main_Render_Kernel << < blockspergrid, threadsperblock >> >(i, bitmap_gpu, cam, wor, steps, random_texture_device, clock(), q_table_device);
    hipEventRecord(stop_kernel);
    gpuErrchk(hipPeekAtLastError());

    //Copy all variables back
    hipMemcpy(camera->getBitmap(), bitmap_gpu, cam.size.y * cam.size.x * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop_kernel);
    hipEventSynchronize(stop);

    float kernelTime, totalTime; // Initialize elapsedTime;
    hipEventElapsedTime(&kernelTime, begin_kernel, stop_kernel);
    hipEventElapsedTime(&totalTime, begin, stop);

    if( (i+=COLUMNS_IN_ONCE) == camera->getWidth())
    {
        i = 0;
        steps++;
        printf("GPU Time: %fms, %fms, steps: %d\n", kernelTime, totalTime -kernelTime, steps);
        camera->incSteps();
//        std::cout<<"Samples Done: "<<camera->getSteps()*SAMPLE*SAMPLE<<std::endl;
        return steps >= 400;
    }
    return false;
}

RenderEngine_GPU::~RenderEngine_GPU() {
    //Free variables
    hipFree(bitmap_gpu);
    hipFree(random_texture_device);
    free(random_texture);
}

__device__ unsigned int get_q_index(float3 r){
    return clamp(static_cast<uint>((floor(r.x) + MAX_COORD) * MAX_COORD * MAX_COORD * 4
                                           + (floor(r.y) + MAX_COORD) * MAX_COORD * 2
                                           + floor(r.z) + MAX_COORD), (uint)0, (uint)MAX_COORD * MAX_COORD * MAX_COORD * 8);
}

#define DEBUG 1
__device__ float3 computeColor(Ray_GPU ray, int &seed, World_GPU wor, float* q_table) {
    float3 c = AMBIENT_COLOR, c_final;

    unsigned int q_index = get_q_index(ray.orig), last_index=0;
    unsigned char sphere = wor.intersectRay(ray);
    for (unsigned char i = 0; i < MAX_LEVEL; i++){
        last_index = q_index;
        q_index = get_q_index(ray.orig);
//        c = make_float3((floor(ray.orig.x) + MAX_COORD)/(MAX_COORD*2), (floor(ray.orig.y) + MAX_COORD)/(MAX_COORD*2), (floor(ray.orig.z) + MAX_COORD)/(MAX_COORD*2));
        if(i==0) c_final = make_float3(q_table[q_index]);
//        break;
        if(sphere^255) {
            c = c*wor.spheres[sphere].col;
            SPHERE_MATERIAL sp_mat = wor.spheres[sphere].material;
            if(sp_mat == LIGHT){
                //light
                q_table[last_index] = q_table[last_index] * (1-ALPHA) + clamp01(length(wor.spheres[sphere].col))*ALPHA;
                break;
            }else if(sp_mat == DIELECTRIC){
                //dielectric
                float eta = wor.spheres[sphere].param;
                float cosTheta = dot(ray.dir, ray.normal);
                bool isInside = cosTheta > 0;
                float nc=1, nnt=isInside?eta/nc:nc/eta;
                float cos2t = 1-nnt*nnt*(1-cosTheta*cosTheta);
                if (cos2t<0){ //TIR
                    ray.dir = normalize(ray.dir - 2 * ray.normal * cosTheta);
                }else{
                    cosTheta = -fabs(cosTheta);
                    float3 refr_dir = normalize(ray.dir * nnt - ray.normal*((isInside?-1:1)*(cosTheta*nnt+sqrt(cos2t))));

                    float a=eta-nc, b=eta+nc, R0=a*a/(b*b), c1 = 1-(isInside?dot(refr_dir, ray.normal):-cosTheta);
                    float Re=R0+(1-R0)*c1*c1*c1*c1*c1,Tr=1-Re,P=.25f+.5f*Re,RP=Re/P,TP=Tr/(1-P);
                    if (Random_GPU(seed) < P) {
                        c = c * RP;
                        ray.dir = normalize(ray.dir - 2 * ray.normal * cosTheta);
                    }
                    else{
                        c = c * TP;
                        ray.dir = refr_dir;
                    }
                }
                sphere = wor.intersectRay(ray);
            }else if(sp_mat == GLOSSY){
                //glossy
                float cosTheta = dot(ray.dir, ray.normal);
                float n = wor.spheres[sphere].param;

                float phi=2*M_PI*Random_GPU(seed), cosAlpha=pow(Random_GPU(seed), 1.f/(n+1)), sineAlpha = sqrt(1-cosAlpha*cosAlpha);
                float rotAngle = 2*(acos(-cosTheta) + acos(cosAlpha) - M_PI/2);

                float3 w = normalize(ray.dir - 2 * ray.normal * cosTheta);
                float3 u = normalize(cross((fabs(w.x)>.1?make_float3(0,1,0):make_float3(1,0,0)),w));
                float3 v = cross(w,u);

                float3 dDirection = u*cos(phi)*sineAlpha + v*sin(phi)*sineAlpha + w*cosAlpha;

                if(dot(dDirection,ray.normal)<0) {
                    float3 k = normalize(cross(w, ray.normal));
                    dDirection = cos(rotAngle) * dDirection + sin(rotAngle) * cross(k, dDirection);
                }
                ray.dir = dDirection;
                sphere = wor.intersectRay(ray);
            }else if(sp_mat == REFLECTIVE && Random_GPU(seed) < wor.spheres[sphere].param){
                float cosTheta = dot(ray.dir, ray.normal);
                ray.dir = normalize(ray.dir - 2 * ray.normal * cosTheta);
                sphere = wor.intersectRay(ray);
            }else {
                //diffuse
                q_table[last_index] = q_table[last_index] * (1-ALPHA) + clamp01(q_table[q_index]*0.8f)*ALPHA;
                float alpha=2*M_PI* Random_GPU(seed),z= Random_GPU(seed), sineTheta = sqrtf(1-z);
                float3 w = ray.normal;
                float3 u = normalize(cross((fabs(w.x)>.1?make_float3(0,1,0):make_float3(1,0,0)),w));
                float3 v = cross(w,u);
                ray.dir = u*cos(alpha)*sineTheta + v*sin(alpha)*sineTheta + w*sqrt(z);
                sphere = wor.intersectRay(ray);

            }
        }else{
//            c = BACKGROUND;
            break;
        }
        if(i^3 && length(c) < 0.07){
            break;
        }
    }
    return DEBUG?c_final:c;
}

__global__ void Main_Render_Kernel(int startI, unsigned char *bitmap, Camera_GPU cam, World_GPU wor, unsigned int steps,
                                   int* rand_tex, int clk, float* q_table) { //j->row, i->column
    // <8,8,12>
    unsigned int p = threadIdx.x;
    unsigned int q = threadIdx.y;

    unsigned int j = (blockIdx.x * blockDim.z + threadIdx.z);
    unsigned int i = startI + j/cam.size.y;
    j %= cam.size.y;

    int seed = 341*q + 253 * p * 8 + ( rand_tex[(i + j*cam.size.x)%(cam.size.x * cam.size.y)]) + 349*steps + clk;
    float _i = i + 1.2f * (p + Random_GPU(seed)) / SAMPLE;
    float _j = j + 1.2f * (q + Random_GPU(seed)) / SAMPLE;

    //Initial Ray direction
    float xw = (1.0f*cam.size.x/cam.size.y * (_i - cam.size.x / 2.0f + 0.5f) / cam.size.x);
    float yw = ((_j - cam.size.y / 2.0f + 0.5f) / cam.size.y);
    float3 dir = normalize(cam.u * xw + cam.v * yw - cam.w * 1.207107f);

    //Create ray
    Ray_GPU ray(cam.pos, dir);
    float3 c = computeColor(ray, seed, wor, q_table);

    c = warpReduceSumTriple(c);
    __shared__ float3 val[MAX_THREADS_IN_BLOCK/(SAMPLE*SAMPLE)];
    val[threadIdx.z] = make_float3(1,0,0);
    __syncthreads();
    if(p==SAMPLE-1 && q==SAMPLE-1)
        val[threadIdx.z] = c;
    __syncthreads();

    if(p==0 && q==0){
        c = c+val[threadIdx.z];
        c = clamp(c/(SAMPLE*SAMPLE), 0, 1);
        int index = (i + j*cam.size.x)*3;
        float f = 1.0f / (steps+1);
        bitmap[index + 0] = (unsigned char) ((bitmap[index + 0] * (f * steps) + 255 * c.x * f));
        bitmap[index + 1] = (unsigned char) ((bitmap[index + 1] * (f * steps) + 255 * c.y * f));
        bitmap[index + 2] = (unsigned char) ((bitmap[index + 2] * (f * steps) + 255 * c.z * f));
    }
}
